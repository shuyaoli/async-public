#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h> // CURAND_RNG_PSEUDO_MTGP32
#include <hiprand/hiprand_kernel.h>
__device__ double atomic_add(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

#define n 4096
#define dim 32
#define s 1
#define epoch 60
#define alpha 0.5
#define SIZE "SMALL"
#define WARP_SIZE 32
#define NUM_THREAD 1024 

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("Message: %s\n", hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)

using namespace std;

void read_var(double* var, string var_name, int len)
{
  string filename = string("../data/") + 
    string(SIZE) + string("/") + var_name + string(".txt");
  ifstream var_file(filename);
  string line;
  if (!var_file.is_open()) {
    cout << "Failed to open " << var_name << endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < len; i++) {
    if (getline(var_file, line)) 
      var[i] = stod(line);
    else {
      cout << "Error loading " << var_name << endl;
      exit(EXIT_FAILURE);
    }
  }
  var_file.close();
}

__global__ void initCurand (hiprandState *states, unsigned long seed) {
  int i = blockIdx.x * blockDim.x + threadIdx.x; // possibly adding time to seq number 
  hiprand_init(seed, i, 0, &states[i]);
}

__global__ void reduction_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_row, int num_col, double div) {
  // Lauch num_col threads in total
  
  // Holds intermediates in shared memory reduction
  __syncthreads();
  __shared__ double buffer[1024/WARP_SIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int lane = threadIdx.x % WARP_SIZE;

  for (int k = 0; k < num_row; k++) {
    int j = (k + blockIdx.x) % num_row;
    //j = k;
    double temp;
    // All threads in a block of 1024 take an element
    temp = z[i + num_col * j];
    
    // All warps in this block (32) compute the sum of all
    // threads in their warp
    for(int delta = WARP_SIZE/2; delta > 0; delta /= 2)
      temp += __shfl_xor_sync(0xffffffff, temp, delta);

    // Write all 32 of these partial sums to shared memory
    if(lane == 0)
      buffer[threadIdx.x / WARP_SIZE] = temp / div;
    
    __syncthreads();

    // Add the remaining 32 partial sums using a single warp
    if(threadIdx.x < WARP_SIZE) {
      temp = buffer[threadIdx.x];
      for(int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
        temp += __shfl_xor_sync(0xffffffff,temp, delta);
    }

    // Add this block's sum to the total sum
    if(threadIdx.x == 0)
      atomic_add(sum_z+j, temp);
    // sum_z[j] += temp;
  }
}


__global__ void parallel_mean_rotate(const double* __restrict__ z,
                                     double* __restrict__ mean_z) {
  // Lauch num_row threads in total
  int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1 ~ dim
  double total = 0;
  for (int c = 0; c < NUM_THREAD; c++) {
    total += z[idx * NUM_THREAD + c];
  }
  mean_z[idx] = total / n;
}

__global__ void zUpdate(const double* __restrict__ x_a,
                        const double* __restrict__ y,
                        double* __restrict__ z_a,
                        const double* __restrict__ mean_z,
                        double* __restrict__ delta_z,
                        hiprandState_t *states)
{

  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int ik =  hiprand (&states[idx]) % n;
  // const int ik = idx % n;

  
  double dot = 0;
  for (int i = 0; i < dim; i++) 
    dot += mean_z[i] * x_a[dim * ik + i];

  for (int c =  0; c < dim; c++) {        
    delta_z[idx+c*NUM_THREAD] = mean_z[c] - z_a[ik + c * n] - 
      alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
  }

  __syncthreads();
  // TODO: lock it!
  for (int c = 0; c < dim; c++) {
    // z_a[ik + c * n] += delta_z[idx + c * NUM_THREAD];
    atomic_add(&z_a[ik + c * n], delta_z[idx + c * NUM_THREAD]);
  }

  // ----UNCOMMENT this loop, then COMMENT OUT everything in the main loop except zUpdate
  // for (int c = 0; c < dim; c++){
  //   atomic_add(&mean_z[c], delta_z[idx + c * NUM_THREAD] / n);
  // }
  //------------------------------------------------------------------------------------
}

int main()
{
  double *x_a = new double [n * dim];
  double *y = new double [n];

  read_var(x_a, "x_a", n * dim);
  read_var(y, "y", n);
  
  double *d_x_a, *d_y;
  CUDA_CALL(hipMalloc(&d_x_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice));


  hiprandState *d_states;
  CUDA_CALL(hipMalloc(&d_states, sizeof(hiprandState) * NUM_THREAD));
  initCurand <<< NUM_THREAD / 1024, 1024 >>> ( d_states, 0);

  
  double* z_a =  new double[n * dim]();
  double* mean_z = new double [dim]();
  double *d_z_a, *d_mean_z;
  CUDA_CALL(hipMalloc(&d_z_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_mean_z, sizeof(double) * dim));                 
  CUDA_CALL(hipMemcpy(d_z_a, z_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));

  double* d_delta_z;
  CUDA_CALL(hipMalloc(&d_delta_z, sizeof(double) * dim * NUM_THREAD));
  //----------SHOULD BE UNNECESSARY--------------------
  double* delta_z = new double[NUM_THREAD * dim]();
  CUDA_CALL(hipMemcpy(d_delta_z, delta_z, sizeof(double) * NUM_THREAD * dim, hipMemcpyHostToDevice));
  //--------------------------------------------------

  double* delta_mean_z = new double [dim]();
  double* d_delta_mean_z;
  CUDA_CALL(hipMalloc(&d_delta_mean_z, sizeof(double) * dim));
  CUDA_CALL(hipMemcpy(d_delta_mean_z, delta_mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
  
  for (int k = 0; k < epoch * n / NUM_THREAD ; k++) { //epoch * n / NUM_THREAD
    // initCurand <<< NUM_THREAD / 1024, 1024 >>> ( d_states, k);
    zUpdate <<< NUM_THREAD / 1024, 1024 >>> (d_x_a, d_y, d_z_a, d_mean_z, d_delta_z, d_states);

    //--------The following code enforce z_mean consistency-----------
    // memset(mean_z, 0, sizeof(double) * dim);
    // CUDA_CALL(cudaMemcpy(d_mean_z, mean_z, sizeof(double) * dim, cudaMemcpyHostToDevice));
    // reduction_sum_divided <<< n / 1024, 1024>>> (d_z_a, d_mean_z, dim, n, n);
    //---------------------------------------------------------------

    //------------------------One way to calculate delta_mean_z-------------------------
    memset(delta_mean_z, 0, sizeof(double) * dim);
    CUDA_CALL(hipMemcpy(d_delta_mean_z, delta_mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
    reduction_sum_divided <<< NUM_THREAD / 1024, 1024>>> (d_delta_z, d_delta_mean_z, dim, NUM_THREAD, n);

    //------------------Another way to calculate delta_mean_z----------------------------
    // parallel_mean_rotate <<< dim / 1024, 1024 >>> (d_delta_z, d_delta_mean_z);
    //---------------------------------------------------------------------------------
    
    CUDA_CALL(hipMemcpy(delta_mean_z, d_delta_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
    
    for (int c = 0; c < dim; c++) {
      mean_z[c] += delta_mean_z[c];
    }
    
    CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
    
  }

  CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < dim; i++) printf("%.15f\n", mean_z[i]);
  
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x_a);
  hipFree(d_y);
  hipFree(d_delta_mean_z);
  hipFree(d_delta_z);
  return 0;
}
