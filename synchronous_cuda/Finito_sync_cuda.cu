#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h> // CURAND_RNG_PSEUDO_MTGP32
#include <hiprand/hiprand_kernel.h>
#include <chrono>

#define WARP_SIZE 32
#define n 16384
#define dim 4096
#define s 1
#define epoch 64
#define alpha 0.5

#define SIZE "HUGE"

#define NUM_PROCESSOR 8192    
#define NUM_AGENT 256

#define UPDATE_BLOCKSIZE 256  
#define SUM_BLOCKSIZE 256
#define MEAN_BLOCKSIZE 256

// zCalculate <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE>>>
// zUpdate    <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE>>>

// parallel_sum_divided <<< dim / SUM_BLOCKSIZE, SUM_BLOCKSIZE>>> (d_delta_z, d_delta_mean_z, NUM_AGENT, dim, n);

// mean_zUpdate <<< dim / MEAN_BLOCKSIZE, MEAN_BLOCKSIZE >>> (d_delta_mean_z, d_mean_z);

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("Message: %s\n", hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("CuRand error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

using namespace std;

// __device__ double atomic_add(double*, double);

//XXX is the __restrict__ keyword legitimate?
//const __restrict__ is valid for x_a and y since they are problem data so they never change
//__restrict__ for mean_z is valid since we do not change (write to) mean_z
//__restrict__ for delta_z is valid since the access is fully separated by indexing for this kernel
//__restrict__ for z_a is XXX not XX valid since (with the atomic writes) different threads can write
//to the same location. (Since there is only one read from z_a, I don't think it will make a difference
//but the __restrict__ seems to be conceptually wrong.)
__global__ void zCalculate(const double* __restrict__ x_a,
                           const double* __restrict__ y,
                           const double* __restrict__ z_a,
                           const double* __restrict__ mean_z,
                           double* __restrict__ delta_z,
                           const unsigned int* __restrict__ random_index,
                           int itr)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int lane = threadIdx.x % WARP_SIZE; // TODO: threadIdx % WARP_SIZE
  const int warpIdx = idx / WARP_SIZE;
  const int ik =  random_index[itr * NUM_AGENT + warpIdx] % n;
  // Coalesced memory access is one of the code optimization
  // considerations in CUDA that actually matters. This has the
  // potential to greatly speed up or slow down your code. Currently,
  // the problem is that adjacent threads access different parts of
  // the dataset with the random number generation. One remedy is to
  // have a single warp access consecutive datapoints (circularly
  // consecutive, so use mod (%) to wrap around) by having only the
  // 0th thread in the warp generate a random index and sharing it
  // among the threads. Another option is to have the 32 threads
  // within a single warp process the same datapoint
  __shared__ double s_mean_z[dim];
 
  for (int c = lane; c < dim; c+=WARP_SIZE)
    s_mean_z[c] = mean_z[c];
  
  __syncwarp();
  //XXX Non-coalesced memory access XXX


  double dot = 0;
  for (int c = lane; c < dim; c+=WARP_SIZE) 
    dot += s_mean_z[c] * x_a[dim * ik + c];

  // Sum up all "dot" in a warp. Store the result in variable "dot" in every thread
  for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
    dot += __shfl_xor_sync(0xffffffff, dot, delta);
  
  //Coalesced memory access (good) for delta_z
  //XXX Non-coalesced memory access for z_a
  //XXX is this for-loop the main bottleneck? XXX
  // Answer: No. It's just one of the bottle neck for now.
  //XXX the read for mean_z could be shared across the block. consider using __shared__ variables
  //XXX or should could read mean_z[c] and share it across the warps using warp-level primitives
  for (int c =  lane; c < dim; c+=WARP_SIZE) {        
    delta_z[warpIdx * dim + c] = s_mean_z[c] - z_a[ik * dim + c] - 
      alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * s_mean_z[c]);
  }
}

__global__ void zUpdate(double* __restrict__ z_a,
                        double* __restrict__ delta_z,
                        unsigned int* __restrict__ random_index,
                        int itr)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int lane = idx % WARP_SIZE; // TODO: threadIdx % WARP_SIZE
  const int warpIdx = idx / WARP_SIZE;
  const int ik =  random_index[itr * NUM_AGENT + warpIdx] % n;
  //XXX Non-coalesced memory access XXX
  for (int c = lane; c < dim; c+=WARP_SIZE) {
    // z_a[ik * dim + c] +=  delta_z[warpIdx * dim + c];
    // atomic gives 50ms performance loss in total for a medium dataset
    
    // int cc = (c + warpIdx * 32) % dim;
    // TODO: This gives only 5ms performance gain; I am not sure why it's so small
    
    atomicAdd(&z_a[ik * dim + c], delta_z[warpIdx * dim + c]); 
  }
}


__global__ void mean_zUpdate (const double* __restrict__ delta_mean_z,
                              double* __restrict__ mean_z) {
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  mean_z[idx] += delta_mean_z[idx];
}

void read_var(double* ,string, int);


__global__ void reduction_sum_divided(const double* __restrict__ z,
                                      double* __restrict__ sum_z,
                                      int num_row, int num_col, double div);


__global__ void parallel_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_row, int num_col, double div);

int main()
{
  double *x_a = new double [n * dim];
  double *y = new double [n];
  
  read_var(x_a, "x_a", n * dim);
  read_var(y, "y", n);

  double *d_x_a, *d_y;
  CUDA_CALL(hipMalloc(&d_x_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice));
  
  double* z_a =  new double[n * dim]();
  double* mean_z = new double [dim]();
  double *d_z_a, *d_mean_z;
  CUDA_CALL(hipMalloc(&d_z_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_mean_z, sizeof(double) * dim));                 
  CUDA_CALL(hipMemcpy(d_z_a, z_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));

  double* d_delta_z;
  CUDA_CALL(hipMalloc(&d_delta_z, sizeof(double) * dim * NUM_AGENT));

  double* delta_mean_z = new double [dim];
  double* d_delta_mean_z;
  CUDA_CALL(hipMalloc(&d_delta_mean_z, sizeof(double) * dim));

  chrono :: duration <double> elapsed (0);

  unsigned int * d_random_index;
  CUDA_CALL(hipMalloc(&d_random_index, sizeof(unsigned int) * n * epoch));
  hiprandGenerator_t gen;
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL)); // seed
  CURAND_CALL(hiprandGenerate(gen, d_random_index, n * epoch));
  
  hipDeviceSynchronize(); auto start = chrono :: high_resolution_clock::now();

  hipStream_t stream1, stream2;
  for (int k = 0; k < epoch * n / NUM_AGENT; k++) {
    memset(delta_mean_z, 0, sizeof(double) * dim);
    CUDA_CALL(hipMemcpy(d_delta_mean_z, delta_mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
    
    zCalculate <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE>>>
      (d_x_a, d_y, d_z_a, d_mean_z, d_delta_z, d_random_index, k);   // 2.6s


    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    zUpdate <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE, 0, stream1>>>
      (d_z_a, d_delta_z, d_random_index, k);
    //--------The following code enforce z_mean consistency somehow inefficiently-----------
    // memset(mean_z, 0, sizeof(double) * dim);
    // CUDA_CALL(cudaMemcpy(d_mean_z, mean_z, sizeof(double) * dim, cudaMemcpyHostToDevice));  // < 0.01s
    // reduction_sum_divided <<< n / 1024, 1024>>> (d_z_a, d_mean_z, dim, n, n); // 0.35 s 
    //---------------------------------------------------------------

    //------------------------One way to calculate delta_mean_z-------------------------

    // reduction_sum_divided <<< NUM_AGENT / 1024, 1024>>>
    //   (d_delta_z, d_delta_mean_z, dim, NUM_AGENT, n); // 0.35 s

    //------------------Another way to calculate delta_mean_z----------------------------
    parallel_sum_divided <<< dim / SUM_BLOCKSIZE, SUM_BLOCKSIZE,0, stream2>>> (d_delta_z, d_delta_mean_z, NUM_AGENT, dim, n);

    //---------------------------------------------------------------------------------

    //---------------Comment out the following code when enforcing z_mean consistency------------

    mean_zUpdate <<< dim / MEAN_BLOCKSIZE, MEAN_BLOCKSIZE, 0, stream2 >>> (d_delta_mean_z, d_mean_z);
    //-------------------------------------------------------------------------------------------

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    
  }
  hipDeviceSynchronize(); auto end = chrono::high_resolution_clock::now(); elapsed += end - start;
  cout << "elapsed time: " << elapsed.count() << " s\n";
  
  CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < 4; i++) printf("%.15f\n", mean_z[i]);
  
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x_a);
  hipFree(d_y);
  hipFree(d_delta_mean_z);
  hipFree(d_delta_z);
  return 0;
}

void read_var(double* var, string var_name, int len)
{
  string filename = string("../data/") + 
    string(SIZE) + string("/") + var_name + string(".txt");
  ifstream var_file(filename);
  string line;
  if (!var_file.is_open()) {
    cout << "Failed to open " << var_name << endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < len; i++) {
    if (getline(var_file, line)) 
      var[i] = stod(line);
    else {
      cout << "Error loading " << var_name << endl;
      exit(EXIT_FAILURE);
    }
  }
  var_file.close();
}

__global__ void reduction_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_row, int num_col, double div) {
  // Lauch num_col threads in total
  
  // Holds intermediates in shared memory reduction
  __syncthreads();
  __shared__ double buffer[1024/WARP_SIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int lane = threadIdx.x % WARP_SIZE;

  for (int k = 0; k < num_row; k++) {
    int j = (k + blockIdx.x) % num_row;
    //j = k;
    double temp;
    // All threads in a block of 1024 take an element
    temp = z[i + num_col * j];
    
    // All warps in this block (32) compute the sum of all
    // threads in their warp
    for(int delta = WARP_SIZE/2; delta > 0; delta /= 2)
      temp += __shfl_xor_sync(0xffffffff, temp, delta);

    // Write all 32 of these partial sums to shared memory
    if(lane == 0)
      buffer[threadIdx.x / WARP_SIZE] = temp / div;
    
    __syncthreads();

    // Add the remaining 32 partial sums using a single warp
    if(threadIdx.x < WARP_SIZE) {
      temp = buffer[threadIdx.x];
      for(int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
        temp += __shfl_xor_sync(0xffffffff,temp, delta);
    }

    // Add this block's sum to the total sum
    if(threadIdx.x == 0)
      atomicAdd(sum_z+j, temp);  
    // sum_z[j] += temp;
  }
}

__global__ void parallel_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_row, int num_col, double div) {
  // Lauch num_col threads in total
  int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1 ~ num_col
  double total = 0;
  for (int c = 0; c < num_row; c++) {
    total += z[idx + c * num_col];
  }
  sum_z[idx] = total / div;
}

// __device__ double atomic_add(double* address, double val)
// {
//   unsigned long long int* address_as_ull =
//     (unsigned long long int*)address;
//   unsigned long long int old = *address_as_ull, assumed;

//   do {
//     assumed = old;
//     old = atomicCAS(address_as_ull, assumed,
//                     __double_as_longlong(val +
//                                          __longlong_as_double(assumed)));

//     // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//   } while (assumed != old);

//   return __longlong_as_double(old);
// }