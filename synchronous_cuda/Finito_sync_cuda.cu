#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h> // CURAND_RNG_PSEUDO_MTGP32
#include <hiprand/hiprand_kernel.h>
#include <chrono>

#define WARP_SIZE 32
#define n 16384
#define dim 2048
#define s 1
#define epoch 64
#define alpha 0.5

#define SIZE "LARGE"

#define NUM_PROCESSOR 16384    // > 1024
#define UPDATE_BLOCKSIZE 512  // <=256

// zUpdate <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE>>>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("Message: %s\n", hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("CuRand error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

using namespace std;

__device__ double atomic_add(double*, double);

__global__ void zUpdate(const double* __restrict__ x_a,
                        const double* __restrict__ y,
                        double* __restrict__ z_a,
                        double* __restrict__ mean_z,
                        double* __restrict__ delta_z,
                        hiprandState_t *states)
                        // unsigned int * random_index,
                        // int itr)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int ik =  hiprand (&states[idx]) % n;
  // const int ik =  random_index [itr * NUM_PROCESSOR + idx] % n;
  // const int ik = idx;
  
  
  double dot = 0;
  for (int i = 0; i < dim; i++) 
    dot += mean_z[i] * x_a[dim * ik + i];

  for (int c =  0; c < dim; c++) {        
    delta_z[idx+c*NUM_PROCESSOR] = mean_z[c] - z_a[ik + c * n] - 
      alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
  }

  for (int c = 0; c < dim; c++) {
    // z_a[ik + c * n] += delta_z[idx + c * NUM_PROCESSOR];
    atomic_add(&z_a[ik + c * n], delta_z[idx + c * NUM_PROCESSOR]); // atomic gives < 0.1s performance loss
  }
}

void read_var(double* ,string, int);

__global__ void initCurand (hiprandState *states, unsigned long seed) {
  int i = blockIdx.x * blockDim.x + threadIdx.x; // possibly adding time to seq number 
  hiprand_init(seed, i, 0, &states[i]);
}

__global__ void reduction_sum_divided(const double* __restrict__ z,
                                      double* __restrict__ sum_z,
                                      int num_row, int num_col, double div);


__global__ void parallel_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_col, double div);

int main()
{
  double *x_a = new double [n * dim];
  double *y = new double [n];
  
  read_var(x_a, "x_a", n * dim);
  read_var(y, "y", n);

  double *d_x_a, *d_y;
  CUDA_CALL(hipMalloc(&d_x_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice));


  hiprandState *d_states;
  CUDA_CALL(hipMalloc(&d_states, sizeof(hiprandState) * NUM_PROCESSOR));
  initCurand <<< NUM_PROCESSOR / 1024, 1024 >>> ( d_states, 0); //TODO: seed with time

  
  double* z_a =  new double[n * dim]();
  double* mean_z = new double [dim]();
  double *d_z_a, *d_mean_z;
  CUDA_CALL(hipMalloc(&d_z_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_mean_z, sizeof(double) * dim));                 
  CUDA_CALL(hipMemcpy(d_z_a, z_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));

  double* d_delta_z;
  CUDA_CALL(hipMalloc(&d_delta_z, sizeof(double) * dim * NUM_PROCESSOR));

  double* delta_mean_z = new double [dim];
  double* d_delta_mean_z;
  CUDA_CALL(hipMalloc(&d_delta_mean_z, sizeof(double) * dim));

  chrono :: duration <double> elapsed (0);

  // unsigned int * d_random_index;
  // CUDA_CALL(cudaMalloc(&d_random_index, sizeof(unsigned int) * n * epoch));
  // curandGenerator_t gen;
  // CURAND_CALL(curandCreateGenerator(&gen, CURAND_RNG_PSEUDO_DEFAULT));
  // CURAND_CALL(curandSetPseudoRandomGeneratorSeed(gen, 1234ULL)); // seed
  // CURAND_CALL(curandGenerate(gen, d_random_index, n * epoch));
  
  hipDeviceSynchronize(); auto start = chrono :: high_resolution_clock::now();
  for (int k = 0; k < epoch * n / NUM_PROCESSOR ; k++) {
    // initCurand <<< NUM_THREAD / 1024, 1024 >>> ( d_states, k);
    
    zUpdate <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE>>>
      (d_x_a, d_y, d_z_a, d_mean_z, d_delta_z, d_states);      // 2.6s
    // zUpdate <<< NUM_PROCESSOR / UPDATE_BLOCKSIZE, UPDATE_BLOCKSIZE>>>
      // (d_x_a, d_y, d_z_a, d_mean_z, d_delta_z, d_random_index, k);
    //--------The following code enforce z_mean consistency somehow inefficiently-----------
    // memset(mean_z, 0, sizeof(double) * dim);
    // CUDA_CALL(cudaMemcpy(d_mean_z, mean_z, sizeof(double) * dim, cudaMemcpyHostToDevice));  // < 0.01s
    // reduction_sum_divided <<< n / 1024, 1024>>> (d_z_a, d_mean_z, dim, n, n); // 0.35 s 
    //---------------------------------------------------------------

    //------------------------One way to calculate delta_mean_z-------------------------
    memset(delta_mean_z, 0, sizeof(double) * dim);
    CUDA_CALL(hipMemcpy(d_delta_mean_z, delta_mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
    // < 0.01 s
    reduction_sum_divided <<< NUM_PROCESSOR / 1024, 1024>>>
      (d_delta_z, d_delta_mean_z, dim, NUM_PROCESSOR, n); // 0.35 s

    //------------------Another way to calculate delta_mean_z----------------------------
    // parallel_sum_divided <<< dim / 1024, 1024 >>> (d_delta_z, d_delta_mean_z, NUM_PROCESSOR, n);
    //---------------------------------------------------------------------------------

    //---------------Comment out the following code when enforcing z_mean consistency------------
    
    CUDA_CALL(hipMemcpy(delta_mean_z, d_delta_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
    
    for (int c = 0; c < dim; c++) {
      mean_z[c] += delta_mean_z[c];
    }
    
    CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
    //-------------------------------------------------------------------------------------------
    // < 0.01 s, including memory transfer time
    
    
  }
  hipDeviceSynchronize(); auto end = chrono::high_resolution_clock::now(); elapsed += end - start;
  cout << "elapsed time: " << elapsed.count() << " s\n";
  
  CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < 4; i++) printf("%.15f\n", mean_z[i]);
  
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x_a);
  hipFree(d_y);
  hipFree(d_delta_mean_z);
  hipFree(d_delta_z);
  hipFree(d_states);
  return 0;
}

void read_var(double* var, string var_name, int len)
{
  string filename = string("../data/") + 
    string(SIZE) + string("/") + var_name + string(".txt");
  ifstream var_file(filename);
  string line;
  if (!var_file.is_open()) {
    cout << "Failed to open " << var_name << endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < len; i++) {
    if (getline(var_file, line)) 
      var[i] = stod(line);
    else {
      cout << "Error loading " << var_name << endl;
      exit(EXIT_FAILURE);
    }
  }
  var_file.close();
}

__global__ void reduction_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_row, int num_col, double div) {
  // Lauch num_col threads in total
  
  // Holds intermediates in shared memory reduction
  __syncthreads();
  __shared__ double buffer[1024/WARP_SIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int lane = threadIdx.x % WARP_SIZE;

  for (int k = 0; k < num_row; k++) {
    int j = (k + blockIdx.x) % num_row;
    //j = k;
    double temp;
    // All threads in a block of 1024 take an element
    temp = z[i + num_col * j];
    
    // All warps in this block (32) compute the sum of all
    // threads in their warp
    for(int delta = WARP_SIZE/2; delta > 0; delta /= 2)
      temp += __shfl_xor_sync(0xffffffff, temp, delta);

    // Write all 32 of these partial sums to shared memory
    if(lane == 0)
      buffer[threadIdx.x / WARP_SIZE] = temp / div;
    
    __syncthreads();

    // Add the remaining 32 partial sums using a single warp
    if(threadIdx.x < WARP_SIZE) {
      temp = buffer[threadIdx.x];
      for(int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
        temp += __shfl_xor_sync(0xffffffff,temp, delta);
    }

    // Add this block's sum to the total sum
    if(threadIdx.x == 0)
      atomic_add(sum_z+j, temp);  
    // sum_z[j] += temp;
  }
}

__global__ void parallel_sum_divided(const double* __restrict__ z,
                                     double* __restrict__ sum_z,
                                     int num_col, double div) {
  // Lauch num_row threads in total
  int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1 ~ num_row
  double total = 0;
  for (int c = 0; c < num_col; c++) {
    total += z[idx * num_col + c];
  }
  sum_z[idx] = total / div;
}

__device__ double atomic_add(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

