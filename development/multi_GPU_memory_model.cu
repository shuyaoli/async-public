# include <iostream>
# include <hip/hip_runtime.h>
# define n 1024
# define dim 32
# include <cassert>
#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      printf("Message: %s\n", hipGetErrorString(x));   \
      exit(1);}} while(0)

using namespace std;

int main()
{
  // Check whether Peer to peer access is available
  hipSetDevice(0);
  int canAccessPeer;
  hipError_t cudaResult;
  cudaResult = hipDeviceCanAccessPeer(&canAccessPeer, 1, 0);
  cout << "Query Success: " << (cudaResult == hipSuccess) << endl;
  cout << "Query Result: " << canAccessPeer << endl;

  // Get device property
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 1);
  cout << prop.canMapHostMemory << endl;

  // Portable memory (page-locked memory)
  double *d_z_a, *d_mean_z, *h_z_a, *h_mean_z;
  CUDA_CALL(hipSetDevice(0));
  CUDA_CALL(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CALL(hipHostAlloc(&h_z_a, sizeof(double) * n * dim, hipHostMallocPortable & hipHostMallocWriteCombined));
  CUDA_CALL(hipHostAlloc(&h_mean_z, sizeof(double) * dim, hipHostMallocPortable));
  
  if (hipHostGetDevicePointer()) {
    CUDA_CALL(hipHostGetDevicePointer(&h_z_a, d_z_a, 0));
    CUDA_CALL(hipHostGetDevicePointer(&h_mean_z, d_mean_z, 0));
  }
 
  hipHostFree(h_z_a);
  hipHostFree(h_mean_z);
  
  unsigned int flags;
  hipGetDeviceFlags(&flags);
  cout << &flags << endl;
  
  return 0;
}
