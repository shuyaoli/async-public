#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include "mex.h"
#include "matrix.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      printf("Message: %s\n", hipGetErrorString(x));   \
      assert(false);}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
      printf("CuRand error at %s:%d\n",__FILE__,__LINE__);      \
      assert(false);}} while(0)

#define WARP_SIZE 32

using namespace std;
using namespace chrono;

__global__ void run_async(const double* __restrict__ x_a,
                          const double* __restrict__ y,
                          double* z_a,
                          double* mean_z,
                          hiprandState* states,
                          int* itr_ptr,
                          int n, int dim, double alpha, double s, int epoch,
                          long long seed)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int lane = threadIdx.x % WARP_SIZE; // TODO: threadIdx % WARP_SIZE
  const int warpIdx = idx / WARP_SIZE;
  double delta_buffer;
  
  if (lane == 0) {
    hiprand_init(seed, warpIdx, 0, &states[warpIdx]);
  }
  
  while (*itr_ptr < epoch * n) {
    int ik;
    if (lane == 0) {
      atomicAdd_system(itr_ptr, 1);
      ik = hiprand(&states[warpIdx]) % n;
    }
    ik = __shfl_sync(0xffffffff, ik, 0);

    double dot = 0;
    for (int c = lane; c < dim; c+=WARP_SIZE) 
      dot += mean_z[c] * x_a[dim * ik + c];
  
    __syncwarp();
    for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
      dot += __shfl_xor_sync(0xffffffff, dot, delta);

    for (int c =  lane; c < dim; c+=WARP_SIZE) {
      // Intention: different warps can start from different coordinates to avoid collision
      // int d = (c + warpIdx * WARP_SIZE) % dim;
      // Doesn't work, basically no speed up.
      delta_buffer = mean_z[c] - z_a[ik * dim + c] - 
        alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
      atomicAdd_system(&z_a[ik * dim + c], delta_buffer);
      atomicAdd_system(&mean_z[c], delta_buffer / n);
    }

    
  }
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
  // prhs[0] and prhs[1] stores x_a and y respectively
  const int n = mxGetDimensions(prhs[1])[0];
  const int dim = mxGetDimensions(prhs[0])[0] /  n;
  
  const double* x_a = mxGetPr(prhs[0]);
  const double* y = mxGetPr(prhs[1]);
  const double alpha = *mxGetPr(prhs[2]);
  const double s = *mxGetPr(prhs[3]);
  const int epoch = *mxGetPr(prhs[4]);
  const int NUM_AGENT = *mxGetPr(prhs[5]);
  const int BLOCKSIZE = *mxGetPr(prhs[6]);

  // should reside on each GPUs
  hipSetDevice(0);
  double *d_x0_a, *d_y0;
  CUDA_CALL(hipMalloc(&d_x0_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y0, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x0_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y0, y, sizeof(double) * n, hipMemcpyHostToDevice));

  hipSetDevice(1);
  double *d_x1_a, *d_y1;
  CUDA_CALL(hipMalloc(&d_x1_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y1, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x1_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y1, y, sizeof(double) * n, hipMemcpyHostToDevice));
  
  // try to reside on host
  double *d_z_a, *d_mean_z;
  CUDA_CALL(hipMallocManaged(&d_z_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMallocManaged(&d_mean_z, sizeof(double) * dim));  
  memset(d_z_a, 0, sizeof(double) * n * dim);
  memset(d_mean_z, 0, sizeof(double) * dim);

  // try to reside on host
  int* d_itr_ptr;
  CUDA_CALL(hipMallocManaged(&d_itr_ptr, sizeof(int)));
  memset(d_itr_ptr, 0, sizeof(int));

  // shouldn't matter, give each GPU a copy for it not to be a bottleneck
  // TODO: taken care of lat
  hipSetDevice(0);
  hiprandState *d_states0;
  CUDA_CALL(hipMalloc(&d_states0, sizeof(hiprandState) * NUM_AGENT / 2)); // TODO: hardcoded

  hipSetDevice(1);
  hiprandState *d_states1;
  CUDA_CALL(hipMalloc(&d_states1, sizeof(hiprandState) * NUM_AGENT / 2));
  
  auto now_clock = time_point_cast<milliseconds>(system_clock::now());
  auto seed = now_clock.time_since_epoch().count();
  
  duration <double> elapsed (0);
  high_resolution_clock :: time_point start, end;
  hipDeviceSynchronize();
  start = high_resolution_clock::now();

  hipSetDevice(0);
  run_async <<< NUM_AGENT * WARP_SIZE / BLOCKSIZE / 2, BLOCKSIZE>>> //TODO hardcoded
    (d_x0_a, d_y0, d_z_a, d_mean_z,  d_states0, d_itr_ptr,
     n, dim, alpha, s, epoch, seed);

  hipSetDevice(1);
  run_async <<< NUM_AGENT * WARP_SIZE / BLOCKSIZE / 2, BLOCKSIZE>>>
    (d_x1_a, d_y1, d_z_a, d_mean_z,  d_states1, d_itr_ptr,
     n, dim, alpha, s, epoch, seed * 2);
  
  hipDeviceSynchronize();
  end = chrono::high_resolution_clock::now();
  elapsed = end - start;
  
  
  // MATLAB Output
  plhs[0] = mxCreateDoubleMatrix(1, dim, mxREAL);
  double * ptr0 = mxGetPr(plhs[0]);
  
  for (int c = 0; c < dim; c++)
    ptr0[c] = d_mean_z[c];
  
  plhs[1] = mxCreateDoubleMatrix(1, 1, mxREAL);
  double *ptr1 = mxGetPr(plhs[1]);
  *ptr1 = elapsed.count();
  
  hipFree(d_itr_ptr);
  hipFree(d_states0);
  hipFree(d_states1);
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x0_a);
  hipFree(d_y0);
  hipFree(d_x1_a);
  hipFree(d_y1);
}