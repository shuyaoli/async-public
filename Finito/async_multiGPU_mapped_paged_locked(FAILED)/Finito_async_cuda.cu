#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#define hipDeviceMapHost 0x08

#define WARP_SIZE 32
#define n 8192
#define dim 1024
#define s 0.1 
#define epoch 64 
#define alpha 4

#define NUM_AGENT 512
#define BLOCKSIZE 128

#define SIZE "MEDIUM"

#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      printf("Message: %s\n", hipGetErrorString(x));   \
      assert(false);}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
      printf("CuRand error at %s:%d\n",__FILE__,__LINE__);      \
      assert(false);}} while(0)

#define WARP_SIZE 32

using namespace std;
using namespace chrono;
void read_var(double* ,string, int);
__global__ void run_async(const double* __restrict__ x_a,
                          const double* __restrict__ y,
                          double* z_a,
                          double* mean_z,
                          hiprandState* states,
                          int* itr_ptr,
                          long long seed)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int lane = threadIdx.x % WARP_SIZE; // TODO: threadIdx % WARP_SIZE
  const int warpIdx = idx / WARP_SIZE;
  double delta_buffer;
  
  if (lane == 0) {
    hiprand_init(seed, warpIdx, 0, &states[warpIdx]);
  }
  
  while (*itr_ptr < epoch * n) {
    int ik;
    if (lane == 0) {
      atomicAdd_system(itr_ptr, 1);
      ik = hiprand(&states[warpIdx]) % n;
    }
    ik = __shfl_sync(0xffffffff, ik, 0);

    double dot = 0;
    for (int c = lane; c < dim; c+=WARP_SIZE) 
      dot += mean_z[c] * x_a[dim * ik + c];
  
    __syncwarp();
    for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
      dot += __shfl_xor_sync(0xffffffff, dot, delta);

    for (int c =  lane; c < dim; c+=WARP_SIZE) {
      // Intention: different warps can start from different coordinates to avoid collision
      // int d = (c + warpIdx * WARP_SIZE) % dim;
      // Doesn't work, basically no speed up.
      delta_buffer = mean_z[c] - z_a[ik * dim + c] - 
        alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
      atomicAdd_system(&z_a[ik * dim + c], delta_buffer);
      atomicAdd_system(&mean_z[c], delta_buffer / n);
    }

    
  }
}

int main()
{
  chrono :: duration <double> elapsed (0);
  chrono :: high_resolution_clock :: time_point start, end;
  
  double *x_a = new double [n * dim];
  double *y = new double [n];
  
  
  read_var(x_a, "x_a", n * dim);
  read_var(y, "y", n);

  // should reside on each GPUs
  hipSetDevice(0);
  double *d_x0_a, *d_y0;
  CUDA_CALL(hipMalloc(&d_x0_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y0, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x0_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y0, y, sizeof(double) * n, hipMemcpyHostToDevice));

  hipSetDevice(1);
  double *d_x1_a, *d_y1;
  CUDA_CALL(hipMalloc(&d_x1_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y1, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x1_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y1, y, sizeof(double) * n, hipMemcpyHostToDevice));
  
  // try to reside on host
  int *h_itr_ptr, *d_itr_ptr;
  CUDA_CALL(hipHostAlloc(&h_itr_ptr, sizeof(int), hipHostMallocMapped));
  CUDA_CALL(hipHostGetDevicePointer(&h_itr_ptr, d_itr_ptr, 0));
  memset(h_itr_ptr, 0, sizeof(int));
  
  double *d_z_a, *d_mean_z, *h_z_a, *h_mean_z;
  CUDA_CALL(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CALL(hipHostAlloc(&h_z_a, sizeof(double) * n * dim, hipHostMallocMapped));
  CUDA_CALL(hipHostAlloc(&h_mean_z, sizeof(double) * dim, hipHostMallocMapped));  
  memset(h_z_a, 0, sizeof(double) * n * dim);
  memset(h_mean_z, 0, sizeof(double) * dim);
  CUDA_CALL(hipHostGetDevicePointer(&h_z_a, d_z_a, 0));
  CUDA_CALL(hipHostGetDevicePointer(&h_mean_z, d_mean_z, 0));

  // shouldn't matter, give each GPU a copy for it not to be a bottleneck
  hipSetDevice(0);
  hiprandState *d_states0;
  CUDA_CALL(hipMalloc(&d_states0, sizeof(hiprandState) * NUM_AGENT / 2)); // TODO: hardcoded

  hipSetDevice(1);
  hiprandState *d_states1;
  CUDA_CALL(hipMalloc(&d_states1, sizeof(hiprandState) * NUM_AGENT / 2));
  
  auto now_clock = time_point_cast<milliseconds>(system_clock::now());
  auto seed = now_clock.time_since_epoch().count();
  
  hipDeviceSynchronize();
  start = high_resolution_clock::now();

  hipSetDevice(0);
  run_async <<< NUM_AGENT * WARP_SIZE / BLOCKSIZE / 2, BLOCKSIZE>>> //TODO hardcoded
    (d_x0_a, d_y0, d_z_a, d_mean_z,  d_states0, d_itr_ptr, seed);

  hipSetDevice(1);
  run_async <<< NUM_AGENT * WARP_SIZE / BLOCKSIZE / 2, BLOCKSIZE>>>
    (d_x1_a, d_y1, d_z_a, d_mean_z,  d_states1, d_itr_ptr, seed * 2);
  
  hipDeviceSynchronize();
  end = chrono::high_resolution_clock::now();
  elapsed = end - start;
  
  for (int i = 0; i < 4; i++) printf("%.15f\n", d_mean_z[i]);
  
  cout <<"NUM_AGENT: " << NUM_AGENT << endl
       <<"BLOCKSIZE: " << BLOCKSIZE << endl
       <<"elapsed time: "<<elapsed.count()<<" s"<<endl
       << endl;
  
  hipFree(d_itr_ptr);
  hipFree(d_states0);
  hipFree(d_states1);
  hipHostFree(h_z_a);
  hipHostFree(h_mean_z);
  hipFree(d_x0_a);
  hipFree(d_y0);
  hipFree(d_x1_a);
  hipFree(d_y1);
  delete []x_a;
  delete []y;
}



void read_var(double* var, string var_name, int len)
{
  string filename = string("../../data/") + 
    string(SIZE) + string("/") + var_name + string(".txt");
  ifstream var_file(filename);
  string line;
  if (!var_file.is_open()) {
    cout << "Failed to open " << var_name << endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < len; i++) {
    if (getline(var_file, line)) 
      var[i] = stod(line);
    else {
      cout << "Error loading " << var_name << endl;
      exit(EXIT_FAILURE);
    }
  }
  var_file.close();
}
