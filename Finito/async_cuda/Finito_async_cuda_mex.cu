#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include "mex.h"
#include "matrix.h"
__device__ double atomic_add(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);
  return __longlong_as_double(old);
}
#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      printf("Message: %s\n", hipGetErrorString(x));   \
      assert(false);}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
      printf("CuRand error at %s:%d\n",__FILE__,__LINE__);      \
      assert(false);}} while(0)

#define WARP_SIZE 32

using namespace std;
using namespace chrono;
// void err_chk(hipError_t err) {
//   if (err != hipSuccess) {
//     cout << hipGetErrorString(err) << endl;
//     assert(false);
//   }
// }
__global__ void run_async(const double* __restrict__ x_a,
                          const double* __restrict__ y,
                          double* z_a,
                          double* mean_z,
                          hiprandState* states,
                          int* itr_ptr,
                          int n, int dim, double alpha, double s, int epoch,
                          long long seed)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int lane = threadIdx.x % WARP_SIZE; // TODO: threadIdx % WARP_SIZE
  const int warpIdx = idx / WARP_SIZE;
  double delta_buffer;
  
  if (lane == 0) {
    hiprand_init(seed, warpIdx, 0, &states[warpIdx]);
  }
  
  while (*itr_ptr < epoch * n) {
    int ik;
    if (lane == 0) {
      atomicAdd(itr_ptr, 1);
      ik = hiprand(&states[warpIdx]) % n;
    }
    ik = __shfl_sync(0xffffffff, ik, 0);

    double dot = 0;
    for (int c = lane; c < dim; c+=WARP_SIZE) 
      dot += mean_z[c] * x_a[dim * ik + c];
  
    __syncwarp();
    for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
      dot += __shfl_xor_sync(0xffffffff, dot, delta);

    for (int c =  lane; c < dim; c+=WARP_SIZE) {
      // int d = (c + warpIdx * WARP_SIZE) % dim;
      // TODO: it doesn't work...basically no speed up
      delta_buffer = mean_z[c] - z_a[ik * dim + c] - 
        alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
      atomicAdd(&z_a[ik * dim + c], delta_buffer);
      atomicAdd(&mean_z[c], delta_buffer / n);
    }
  }
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
  // const char * const errId = "Finito_async_cuda_mex:InvalidInput";
  // const char * const errMsg = "Invalid input to MEX file.";

  // prhs[0] and prhs[1] stores x_a and y respectively
  const int n = mxGetDimensions(prhs[1])[0];
  const int dim = mxGetDimensions(prhs[0])[0] /  n;
  
  const double* x_a = mxGetPr(prhs[0]);
  const double* y = mxGetPr(prhs[1]);
  const double alpha = *mxGetPr(prhs[2]);
  const double s = *mxGetPr(prhs[3]);
  const int epoch = *mxGetPr(prhs[4]);
  const int NUM_AGENT = *mxGetPr(prhs[5]);
  const int BLOCKSIZE = *mxGetPr(prhs[6]);
  
  double *d_x_a, *d_y;
  CUDA_CALL(hipMalloc(&d_x_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice));
  
  double* z_a =  new double[n * dim]();
  double* mean_z = new double [dim]();
  double *d_z_a, *d_mean_z;
  CUDA_CALL(hipMalloc(&d_z_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_mean_z, sizeof(double) * dim));                 
  CUDA_CALL(hipMemcpy(d_z_a, z_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));

  int zero = 0;
  int* d_itr_ptr;
  CUDA_CALL(hipMalloc(&d_itr_ptr, sizeof(int)));
  CUDA_CALL(hipMemcpy(d_itr_ptr, &zero, sizeof(int), hipMemcpyHostToDevice));

  hiprandState *d_states;
  CUDA_CALL(hipMalloc(&d_states, sizeof(hiprandState) * NUM_AGENT));
  auto now_clock = time_point_cast<milliseconds>(system_clock::now());
  auto seed = now_clock.time_since_epoch().count();
  
  duration <double> elapsed (0);
  high_resolution_clock :: time_point start, end;

  hipDeviceSynchronize();
  start = high_resolution_clock::now();
  
  run_async <<< NUM_AGENT * WARP_SIZE / BLOCKSIZE, BLOCKSIZE>>>
    (d_x_a, d_y, d_z_a, d_mean_z,  d_states, d_itr_ptr,
     n, dim, alpha, s, epoch, seed);
  
  hipDeviceSynchronize();
  end = chrono::high_resolution_clock::now();
  elapsed = end - start;
  
  CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
  
  // cout<< endl
  //     <<"NUM_AGENT: " << NUM_AGENT << endl
  //     <<"BLOCKSIZE: " << BLOCKSIZE << endl
  //     <<"elapsed time: "<<elapsed.count()<<" s"<<endl
  //     << endl;
  
  // MATLAB Output
  plhs[0] = mxCreateDoubleMatrix(1, dim, mxREAL);
  double * ptr0 = mxGetPr(plhs[0]);
  
  for (int c = 0; c < dim; c++)
    ptr0[c] = mean_z[c];
  
  plhs[1] = mxCreateDoubleMatrix(1, 1, mxREAL);
  double *ptr1 = mxGetPr(plhs[1]);
  *ptr1 = elapsed.count();
  
  hipFree(d_states);
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x_a);
  hipFree(d_y);
  delete []mean_z;
  delete []z_a;
}