#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand.h> // HIPRAND_RNG_PSEUDO_MTGP32
#include <hiprand/hiprand_kernel.h>
__device__ double atomic_add(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

#define n 4096
#define dim 32
#define s 1
#define epoch 200
#define alpha 0.5
#define SIZE "SMALL"
#define WARP_SIZE 32
#define NUM_THREAD 8

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("Message: %s\n", hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)

using namespace std;

void err_chk(hipError_t err) {
  if (err != hipSuccess) {
    cout << hipGetErrorString(err) << endl;
    assert(false);
  }
}

void read_var(double* var, string var_name, int len)
{
  string filename =
    string(SIZE) + string("/") + var_name + string(".txt");
  ifstream var_file(filename);
  string line;
  if (!var_file.is_open()) {
    cout << "Failed to open " << var_name << endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < len; i++) {
    if (getline(var_file, line)) 
      var[i] = stod(line);
    else {
      cout << "Error loading " << var_name << endl;
      exit(EXIT_FAILURE);
    }
  }
  var_file.close();
}
// randomness design choice:
// different threads have different seeds; the same thread across different kernel lauches have the same seed but different sequence number
// 

__global__ void initCurand (hiprandState *states, unsigned long seed) {
  int i = blockIdx.x * blockDim.x + threadIdx.x; // possibly adding time to seq number 
  hiprand_init(seed, i, 0, &states[i]);
}

__global__ void parallel_sum(const double* __restrict__ z,
                             double *sum_z) {
  //Holds intermediates in shared memory reduction
  __syncthreads();
  __shared__ double buffer[1024/WARP_SIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int lane = threadIdx.x % WARP_SIZE;

  for (int k=0; k<dim; k++) {
    int j = (k + blockIdx.x) % dim;
    //j = k;
    double temp;
    // All threads in a block of 1024 take an element
    temp = z[i + n*j];
    
    // All warps in this block (32) compute the sum of all
    // threads in their warp
    for(int delta = WARP_SIZE/2; delta > 0; delta /= 2)
      temp += __shfl_xor_sync(0xffffffff, temp, delta);

    // Write all 32 of these partial sums to shared memory
    if(lane == 0)
      buffer[threadIdx.x / WARP_SIZE] = temp / n;
    
    __syncthreads();

    // Add the remaining 32 partial sums using a single warp
    if(threadIdx.x < WARP_SIZE) {
      temp = buffer[threadIdx.x];
      for(int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
        temp += __shfl_xor_sync(0xffffffff,temp, delta);
    }

    // Add this block's sum to the total sum
    if(threadIdx.x == 0)
      atomic_add(sum_z+j, temp);
    // sum_z[j] += temp;
  }
}

__global__ void zUpdate(const double* __restrict__ x_a,
                        const double* __restrict__ y,
                        double* z_a,
                        const double* __restrict__ mean_z,
                        double*  delta_z,
                        hiprandState_t *states)
{
  // const int ik = blockDim.x*blockIdx.x + threadIdx.x;
  const int idx = blockDim.x*blockIdx.x + threadIdx.x;
  const int ik =  hiprand (&states[idx]) % n;

  double dot = 0;
  for (int i = 0; i < dim; i++) 
    dot += mean_z[i] * x_a[dim * ik + i];

  for (int c =  0; c < dim; c++) {        
    delta_z[idx+c*NUM_THREAD] = mean_z[c] - z[ik + c * n] - 
      alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
  }

  // TODO: lock it!
  for (int c = 0; c < dim; c++) {
    z[ik + c * n] += delta_z[idx + c * NUM_THREAD];
  }
  
}

int main()
{
  double *x_a = new double [n * dim];
  double *y = new double [n];

  read_var(x_a, "x_a", n * dim);
  read_var(y, "y", n);
  
  double *d_x_a, *d_y;
  CUDA_CALL(hipMalloc(&d_x_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_y, sizeof(double) * n ));
  CUDA_CALL(hipMemcpy(d_x_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice));


  hiprandState *d_states;
  CUDA_CALL(hipMalloc(&d_states, sizeof(hiprandState) * n));
  initCurand <<< n / 1024, 1024 >>> ( d_states, 0);

  
  double* z_a =  new double[n * dim]();
  double* mean_z = new double [dim]();
  double *d_z_a, *d_mean_z;
  CUDA_CALL(hipMalloc(&d_z_a, sizeof(double) * n * dim));
  CUDA_CALL(hipMalloc(&d_mean_z, sizeof(double) * dim));                 
  CUDA_CALL(hipMemcpy(d_z_a, z_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
  
  double* d_delta_z;
  CUDA_CALL(hipMalloc(&d_delta_z, sizeof(double) * dim * n));
  

  for (int k = 0; k < epoch; k++) {
    // initCurand <<< n / 1024, 1024 >>> ( d_states, k);
    zUpdate <<< n / 1024, 1024 >>> (d_x_a, d_y, d_z_a, d_mean_z, d_delta_z, d_states);


    memset(mean_z, 0, sizeof(double) * dim);
    CUDA_CALL(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));

    
    parallel_sum <<< n / 1024, 1024>>> (d_z_a, d_mean_z);
  }

  CUDA_CALL(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
  
  for (int i = 0; i < dim; i++) printf("%.15f\n", mean_z[i]);
  
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x_a);
  hipFree(d_y);
  return 0;
}
