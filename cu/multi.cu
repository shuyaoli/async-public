# include <iostream>
# include <hip/hip_runtime.h>

using namespace std;

int main()
{
  hipSetDevice(0);
  int canAccessPeer;
  hipError_t cudaResult;
  cudaResult = hipDeviceCanAccessPeer(&canAccessPeer, 1, 0);
  cout << "Query Success: " << (cudaResult == hipSuccess) << endl;
  cout << "Query Result: " << canAccessPeer << endl;
  return 0;
}