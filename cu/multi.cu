# include <iostream>
# include <hip/hip_runtime.h>

using namespace std;

int main()
{
  hipSetDevice(0);
  int canAccessPeer;
  hipError_t cudaResult;
  cudaResult = hipDeviceCanAccessPeer(&canAccessPeer, 1, 0);
  cout << "Query Success: " << (cudaResult == hipSuccess) << endl;
  cout << "Query Result: " << canAccessPeer << endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 3);
  cout << prop.unifiedAddressing << endl;
  return 0;
}