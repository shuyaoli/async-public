#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
__device__ double atomic_add(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

#define n 4096
#define dim 32
#define s 1
#define epoch 40
#define alpha 0.5
#define SIZE "SMALL"
#define WARP_SIZE 32

using namespace std;

void err_chk(hipError_t err) {
  if (err != hipSuccess) {
    cout << hipGetErrorString(err) << endl;
    assert(false);
  }
}

void read_var(double* var, string var_name, int len)
{
  string filename = string("../data/") + 
    string(SIZE) + string("/") + var_name + string(".txt");
  ifstream var_file(filename);
  string line;
  if (!var_file.is_open()) {
    cout << "Failed to open " << var_name << endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < len; i++) {
    if (getline(var_file, line)) 
      var[i] = stod(line);
    else {
      cout << "Error loading " << var_name << endl;
      exit(EXIT_FAILURE);
    }
  }
  var_file.close();
}

__global__ void parallel_sum(const double* __restrict__ z,
                             double *sum_z) {
  //Holds intermediates in shared memory reduction
  __syncthreads();
  __shared__ double buffer[1024/WARP_SIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int lane = threadIdx.x % WARP_SIZE;

  for (int k=0; k<dim; k++) {
    int j = (k + blockIdx.x) % dim;
    //j = k;
    double temp;
    // All threads in a block of 1024 take an element
    temp = z[i + n*j];
    
    // All warps in this block (32) compute the sum of all
    // threads in their warp
    for(int delta = WARP_SIZE/2; delta > 0; delta /= 2)
      temp += __shfl_xor_sync(0xffffffff, temp, delta);

    // Write all 32 of these partial sums to shared memory
    if(lane == 0)
      buffer[threadIdx.x / WARP_SIZE] = temp / n;
    
    __syncthreads();

    // Add the remaining 32 partial sums using a single warp
    if(threadIdx.x < WARP_SIZE) {
      temp = buffer[threadIdx.x];
      for(int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
        temp += __shfl_xor_sync(0xffffffff,temp, delta);
    }

    // Add this block's sum to the total sum
    if(threadIdx.x == 0)
        atomic_add(sum_z+j, temp);
      // sum_z[j] += temp;
  }
}

__global__ void zUpdate(const double* __restrict__ x_a,
                        const double* __restrict__ y,
                        double* z_a,
                        const double* __restrict__ mean_z)
{
  const int ik = blockDim.x*blockIdx.x + threadIdx.x;
  
      
  double dot = 0;
  for (int i = 0; i < dim; i++) 
    dot += mean_z[i] * x_a[dim * ik + i];

  for (int c =  0; c < dim; c++) {        
    z_a[ik+c*n] = mean_z[c] -
      alpha * (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
  }
  
}

int main()
{
  double *x_a = new double [n * dim];
  double *y = new double [n];

  read_var(x_a, "x_a", n * dim);
  read_var(y, "y", n);
  
  double* z_a =  new double[n * dim]();

  double* mean_z = new double [dim]();

  double *d_x_a, *d_y;
  double *d_z_a, *d_mean_z;

  err_chk(hipMalloc(&d_x_a, sizeof(double) * n * dim));
  err_chk(hipMalloc(&d_y, sizeof(double) * n ));
  err_chk(hipMalloc(&d_z_a, sizeof(double) * n * dim));
  err_chk(hipMalloc(&d_mean_z, sizeof(double) * dim));

  err_chk(hipMemcpy(d_x_a, x_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  err_chk(hipMemcpy(d_y, y, sizeof(double) * n, hipMemcpyHostToDevice));
  err_chk(hipMemcpy(d_z_a, z_a, sizeof(double) * n * dim, hipMemcpyHostToDevice));
  err_chk(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
  
  for (int k = 0; k < epoch; k++) {
    
    zUpdate <<< n / 1024, 1024 >>> (d_x_a, d_y, d_z_a, d_mean_z);
    // for (int ik = 0; ik < n; ik++) {
    //     double dot = 0;
    //     for (int i = 0; i < dim; i++) 
    //       dot += mean_z[i] * x_a[dim * ik + i];
        
    //     for (int c =  0; c < dim; c++) {        
    //       z_a[ik+c*n] = mean_z[c] - alpha *
    //         (-1.0 / (1+exp(y[ik] * dot)) * y[ik] * x_a[dim * ik + c] + s * mean_z[c]);
    //     }
    // }

    
    memset(mean_z, 0, sizeof(double) * dim);
    err_chk(hipMemcpy(d_mean_z, mean_z, sizeof(double) * dim, hipMemcpyHostToDevice));
    // err_chk(cudaMemcpy(d_z_a, z_a, sizeof(double) * n * dim, cudaMemcpyHostToDevice));
    parallel_sum <<< n / 1024, 1024>>> (d_z_a, d_mean_z);

    // err_chk(cudaMemcpy(mean_z, d_mean_z, sizeof(double) * dim, cudaMemcpyDeviceToHost));
    
    // for (int c = 0; c < dim; c++) mean_z[c] /= n;

    // err_chk(cudaMemcpy(d_mean_z, mean_z, sizeof(double) * dim, cudaMemcpyHostToDevice));

    // for (int c = 0; c < dim; c++) {
    //   double total = 0;
    //   for (int r = 0; r < n; r++) {
    //     total += z_a[r + c * n];
    //   }
    //   mean_z[c] = total / n;
    // }

  }

  err_chk(hipMemcpy(mean_z, d_mean_z, sizeof(double) * dim, hipMemcpyDeviceToHost));
  for (int i = 0; i < dim; i++) printf("%.15f\n", mean_z[i]);
  hipFree(d_z_a);
  hipFree(d_mean_z);
  hipFree(d_x_a);
  hipFree(d_y);
  return 0;
}
